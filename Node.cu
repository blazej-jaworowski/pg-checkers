#include "hip/hip_runtime.h"
#include "Node.cuh"

#include <thrust/reduce.h>

void Node::simulation_step_cpu(Node *node, int game_count)
{
    int result = 0;
    for (int i = 0; i < game_count; i++)
    {
        result += node->game_state.simulate_game();
    }
    node->propagate_result(result, game_count * 2);
}

void run_simulation_step_0(GameState game_state, uint8_t * results, int game_count) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index > game_count) return;

    results[index] = game_state.simulate_game();
}

template<void(*F)(GameState, uint8_t*, int)>
void Node::simulation_step_gpu<F>(Node *node, int game_count)
{
    uint8_t * results;
    hipMalloc(&results, game_count);
    F<<<game_count / 1024 + 1, 1024>>>(node->game_state, results, game_count);
    int result = thrust::reduce(results, results + game_count, 0);
    node->propagate_result(result, game_count * 2);
}

void Node::propagate_result(int white_points, int max_points)
{
    points += game_state.black_turn ? max_points - white_points : white_points;
    total_points += max_points;
    if (parent != nullptr)
        parent->propagate_result(white_points, max_points);
}

void Node::expand()
{
    int empty_index = rand() % (game_state.valid_move_count - child_count);
    for (int i = 0; i < game_state.valid_move_count; i++)
    {
        if (children[i] != nullptr)
            continue;

        if (empty_index-- == 0)
            children[i] = new Node(this, game_state.valid_moves[i]);
    }
    child_count++;
    if (child_count == game_state.valid_move_count)
        leaf = false;
}

Node::~Node()
{
    for (int i = 0; i < game_state.valid_move_count; i++)
    {
        if (children[i] == nullptr)
            continue;
        delete children[i];
    }
}

Node::Node(void (*simulation_step)(Node *, int), int game_count) : simulation_step(simulation_step), game_count(game_count), game_state(), parent(nullptr)
{
    memset(children, 0, sizeof(children));
    game_state.calculate_game_state();
    simulation_step(this, game_count);
}

Node::Node(Node *parent, uint16_t move) : simulation_step(parent->simulation_step), game_count(parent->game_count), game_state(parent->game_state), parent(parent), move(move)
{
    memset(children, 0, sizeof(children));
    this->game_state.play_move(move);
    this->game_state.calculate_game_state();
    simulation_step(this, game_count);
}

void Node::step()
{
    if (game_state.finished)
    {
        propagate_result(game_state.result, 2);
        return;
    }
    if (leaf)
    {
        expand();
        return;
    }
    choose_child()->step();
}

Node *Node::choose_child()
{
    Node *r = nullptr;
    double max = -1;
    for (int i = 0; i < game_state.valid_move_count; i++)
    {
        double v = 1.0 * children[i]->points / children[i]->total_points +
                   M_SQRT2 * sqrt(log(1.0 * total_points / 2) / children[i]->total_points);

        if (v > max)
        {
            r = children[i];
            max = v;
        }
    }
    return r;
}

int Node::best_child_index()
{
    bool child_enemy;
    double curr = -1;
    int r = 0;
    for (int i = 0; i < game_state.valid_move_count; i++)
    {
        child_enemy = game_state.black_turn != children[i]->game_state.black_turn;
        double v = 1.0 * (child_enemy ? children[i]->total_points - children[i]->points : children[i]->points) / children[i]->total_points;
        if (v > curr)
        {
            curr = v;
            r = i;
        }
    }
    return r;
}

int Node::get_move_index(uint16_t child_move)
{
    for (int i = 0; i < game_state.valid_move_count; i++)
    {
        if (game_state.valid_moves[i] == child_move)
            return i;
    }
    return -1;
}
