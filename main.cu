#include "hip/hip_runtime.h"
#include <iostream>
#include "Window.cuh"
#include "Logger.cuh"

int play_one_game(Engine black, Engine white)
{
    GameState gs;
    while (!gs.finished)
    {
        uint16_t move = gs.black_turn ? black.get_move() : white.get_move();
        black.play_move(move);
        white.play_move(move);
        gs.play_move(move);
        gs.calculate_game_state();
    }
    return gs.result;
}

float play_games(const Engine &e1, const Engine &e2, int game_count = 200)
{
    int points = 0;

    auto start = std::chrono::high_resolution_clock::now();
    points += rand() % 2 ? 2 - play_one_game(e1, e2) : play_one_game(e2, e1);
    int time = std::chrono::duration_cast<std::chrono::seconds>(
                   std::chrono::high_resolution_clock::now() - start)
                   .count();
    time *= game_count - 1;
    int hours = time / 3600;
    int minutes = (time - hours * 3600) / 60;
    int seconds = (time - hours * 3600 - minutes * 60);
    std::cout << "Expected time: " << hours << "h " << minutes << "m " << seconds << "s\n";

    for (int i = 1; i < game_count; i++)
    {
        points += rand() % 2 ? 2 - play_one_game(e1, e2) : play_one_game(e2, e1);
        if (i % 5 == 4)
            std::cout << '.' << std::flush;
    }
    std::cout << std::endl;
    return 0.5f * points / game_count;
}

void test_method(void (*method)(GameState, uint8_t *, int), int game_count)
{
    GameState gs;
    uint8_t *res;
    hipMalloc(&res, game_count);

    auto start = std::chrono::high_resolution_clock::now();

    hipMemset(res, 0, game_count);
    hipDeviceSynchronize();

    auto time = std::chrono::high_resolution_clock::now();
    auto time_memset = std::chrono::duration_cast<std::chrono::nanoseconds>(
                           time - start)
                           .count();
    start = time;

    method<<<(game_count - 1) / 1024 + 1, 1024>>>(gs, res, game_count);
    hipDeviceSynchronize();

    time = std::chrono::high_resolution_clock::now();
    auto time_method = std::chrono::duration_cast<std::chrono::nanoseconds>(
                           time - start)
                           .count();
    start = time;

    int result = thrust::reduce(thrust::device, res, res + game_count, 0);
    hipDeviceSynchronize();

    time = std::chrono::high_resolution_clock::now();
    auto time_reduce = std::chrono::duration_cast<std::chrono::nanoseconds>(
                           time - start)
                           .count();
    std::string method_name;
    if (method == run_simulation_step_0)
    {
        method_name = "run_simulation_step_0";
    }
    else if (method == run_simulation_step_1)
    {
        method_name = "run_simulation_step_1";
    }
    Logger::save_record(method_name, game_count, time_memset, time_method, time_reduce);

    hipFree(res);
}

void run_tests()
{
    for (int j = 0; j < 10; j++)
    {
        for (int game_count = 1024; game_count <= 1024 * 1024; game_count *= 2)
        {
            if (j % 2)
            {
                test_method(run_simulation_step_1, game_count);
                test_method(run_simulation_step_0, game_count);
            }
            else
            {
                test_method(run_simulation_step_0, game_count);
                test_method(run_simulation_step_1, game_count);
            }

            auto start = std::chrono::high_resolution_clock::now();
            for (int i = 0; i < game_count; i++)
            {
                GameState().simulate_game();
            }
            auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(
                            std::chrono::high_resolution_clock::now() - start)
                            .count();
            Logger::save_record("cpu", game_count, 0, time, 0);
        }
    }
}

int main(int argc, char **argv)
{
    if (argc >= 3)
    {
        long seed = time(nullptr);
        srand(seed);

        Window window(512, 512);
        window.set_players(strcmp(argv[1], "cpu") == 0, strcmp(argv[2], "cpu") == 0);

        window.run();

        return EXIT_SUCCESS;
    }

    Logger::init("log", "v3");
    run_tests();
    return EXIT_SUCCESS;
}
